#ifndef BC_GPU_IMPL
#define BC_GPU_IMPL

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <limits>
#include <cstddef>

namespace BC {
namespace gpu_impl {

template<typename T, typename J> __global__
static void fill(T t, const J j, int sz) {
	for (int i = 0; i < sz; ++i) {
		t[i] = j;
	}
}

template<typename T, typename J> __global__
static void set_heap(T *t, J *j) {
	&t = &j;
}

template<typename T> __global__
static void scalarONE(T *t) {
	*t = 1;
}
template<typename T, typename J> __global__
static void set_stack(T *t, J j) {
	*t = j;
}

template<typename T, typename J> __global__
static void fill(T* t, const J* j, int sz) {
	for (int i = 0; i < sz; ++i) {
		t[i] = j[i];
	}
}
template<typename T, typename J> __global__
static void eval(T* t, const J* j, int sz) {
	for (int i = 0; i < sz; ++i) {
		t[i];
	}
}

template<typename T> __global__
static void zero(T& t, int sz) {
	for (int i = 0; i < sz; ++i) {
		t[i] = 0;
	}
}

template<class T, class J> __global__
static void copy(T t, J j, int sz) {
	for (int i = 0; i < sz; ++i) {
		t[i] = j[i];
	}
}

template<class T>
struct  _max {
	static constexpr T value = std::numeric_limits<T>::max();
};

template<typename T, typename J> __global__
static void randomize(T t, J lower_bound, J upper_bound, int sz, int seed) {

	 hiprandState_t state;
	  hiprand_init(seed, /* the seed controls the sequence of random values that are produced */
	              seed, /* the sequence number is only important with multiple cores */
	              1, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
	              &state);


	for (int i = 0; i < sz; ++i) {
		t[i] = hiprand(&state);
		t[i] /= 10000000000; //curand max value
		t[i] *= (upper_bound - lower_bound);
		t[i] += lower_bound;
	}
}

}


}

#endif
